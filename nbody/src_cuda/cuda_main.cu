/* nbody.c */

#include "nbody.h"
#include "cudaErr.h"
#include "timer.h"
extern void Particle_array_calculate_forces(
	Particle* this_particle_array, 
	Particle *output_array, 
	int number_of_particles, 
	float time_interval , 
	int timestep);

double wall_time ( )
{
	struct timespec ts;
	double res;
	
	clock_gettime(CLOCK_MONOTONIC,&ts);
	
	res = (double) (ts.tv_sec)  + (double) ts.tv_nsec * 1.0e-9;
	
	return res;
}

int main (int argc, char** argv)
{ 
	if (argc != 4) {
		fprintf(stderr,
				"usage: %s <input file> <particle number> <iteration number>\n",
				argv[0]);
		exit(-1);
	}

	Particle* particle_array = (Particle*) NULL;
	Particle* particle_array2 = (Particle*) NULL;
	int timestep;

	FILE *input_data = fopen(argv[1], "r");
	Particle_input_arguments(input_data);
	
	number_of_timesteps = atoi(argv[3]);
	printf("executing the program %d iterations \n", number_of_timesteps);

	particle_array = Particle_array_construct(number_of_particles);
	particle_array2 = Particle_array_construct(number_of_particles);

	Particle_array_initialize(particle_array, number_of_particles);

	FILE * fileptr = fopen("nbody_out.xyz", "w");
	Particle_array_output_xyz(fileptr, particle_array, number_of_particles);
	
	if (number_of_particles <= 1) {
		printf("Returning\n");		
		return 0;
	}
	
	Particle_array_initialize(particle_array, number_of_particles);

    double makespan = 0;

	for (timestep = 1; timestep <= number_of_timesteps; timestep++) {
		// if ((timestep % timesteps_between_outputs) == 0 ) fprintf(stderr, "Starting timestep #%d.\n", timestep);
		START_TIMER;

		Particle_array_calculate_forces(particle_array, particle_array2, number_of_particles, time_interval , timestep);
	    gpuErrchk (hipDeviceSynchronize());

		END_TIMER;
		makespan += TIMER;
		Particle * tmp = particle_array;
		particle_array = particle_array2;
		particle_array2 = tmp;
	}

    printf("%g ms passed\n", makespan);

	if ((number_of_timesteps % timesteps_between_outputs) != 0) {
		Particle_array_output_xyz(fileptr, particle_array, number_of_particles);
	}
	
	Particle_array_output_xyz(fileptr, particle_array, number_of_particles);

	particle_array = Particle_array_destruct(particle_array, number_of_particles);
	
	if (fclose(fileptr) != 0) {
		fprintf(stderr, "ERROR: can't close the output file.\n");
		exit(program_failure_code);
	}
	
	return program_success_code;
}
