#include "hip/hip_runtime.h"
#include "../../comms.h"
#include "../../cuda/shared.h"
#include "../../params.h"
#include "../../shared.h"
#include "../../shared_data.h"
#include "../neutral_interface.h"
#include "neutral.h"
#include "neutral.k"
#include <assert.h>
#include <float.h>
#include <math.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>

// Performs a solve of dependent variables for particle transport.
void solve_transport_2d(
    const int nx, const int ny, const int global_nx, const int global_ny,
    const uint64_t master_key, const int pad, const int x_off, const int y_off,
    const double dt, const int nparticles_total, int* nlocal_particles,
    const int* neighbours, Particle* particles, const double* density,
    const double* edgex, const double* edgey, const double* edgedx,
    const double* edgedy, CrossSection* cs_scatter_table,
    CrossSection* cs_absorb_table, double* energy_deposition_tally,
    uint64_t* nfacets_reduce_array, uint64_t* ncollisions_reduce_array,
    uint64_t* nprocessed_reduce_array, uint64_t* facet_events,
    uint64_t* collision_events) {

  // This is the known starting number of particles
  int nparticles = *nlocal_particles;
  int nparticles_sent[NNEIGHBOURS];

  if (!nparticles) {
    printf("Out of particles\n");
    return;
  }

  handle_particles(
      global_nx, global_ny, nx, ny, master_key, pad, x_off, y_off, 1, dt,
      neighbours, density, edgex, edgey, edgedx, edgedy, facet_events,
      collision_events, nparticles_sent, nparticles_total, nparticles,
      particles, cs_scatter_table, cs_absorb_table, energy_deposition_tally,
      nfacets_reduce_array, ncollisions_reduce_array, nprocessed_reduce_array);
}

// Handles the current active batch of particles
void handle_particles(
    const int global_nx, const int global_ny, const int nx, const int ny,
    const uint64_t master_key, const int pad, const int x_off, const int y_off,
    const int initial, const double dt, const int* neighbours,
    const double* density, const double* edgex, const double* edgey,
    const double* edgedx, const double* edgedy, uint64_t* facets,
    uint64_t* collisions, int* nparticles_sent, const int nparticles_total,
    const int nparticles_to_process, Particle* particles,
    CrossSection* cs_scatter_table, CrossSection* cs_absorb_table,
    double* energy_deposition_tally, uint64_t* nfacets_reduce_array,
    uint64_t* ncollisions_reduce_array, uint64_t* nprocessed_reduce_array) {

  const int nthreads = NTHREADS;
  const int nblocks = ceil(nparticles_total / (double)NTHREADS);
  handle_particles_kernel<<<nblocks, nthreads>>>(
      nparticles_total, global_nx, global_ny, nx, ny, master_key, pad, x_off,
      y_off, dt, initial, nparticles_total, density, edgex, edgey, edgedx,
      edgedy, energy_deposition_tally, particles->cellx, particles->celly,
      cs_scatter_table->nentries, cs_absorb_table->nentries,
      cs_scatter_table->keys, cs_scatter_table->values, cs_absorb_table->keys,
      cs_absorb_table->values, particles->energy, particles->dt_to_census,
      particles->mfp_to_collision, particles->weight, particles->omega_x,
      particles->omega_y, particles->x, particles->y, nfacets_reduce_array,
      ncollisions_reduce_array, nprocessed_reduce_array);

  // Finalise the reduction of the balance tallies
  uint64_t nfacets = 0;
  uint64_t ncollisions = 0;
  uint64_t nprocessed = 0;
  finish_sum_uint64_reduce(nblocks, nfacets_reduce_array, &nfacets);
  finish_sum_uint64_reduce(nblocks, ncollisions_reduce_array, &ncollisions);
  finish_sum_uint64_reduce(nblocks, nprocessed_reduce_array, &nprocessed);

  *facets = nfacets;
  *collisions = ncollisions;

  printf("Particles  %llu\n", nprocessed);
}

// Initialises a new particle ready for tracking
size_t inject_particles(const int nparticles, const int global_nx,
                        const int local_nx, const int local_ny, const int pad,
                        const double local_particle_left_off,
                        const double local_particle_bottom_off,
                        const double local_particle_width,
                        const double local_particle_height, const int x_off,
                        const int y_off, const double dt, const double* edgex,
                        const double* edgey, const double initial_energy,
                        Particle** particles) {

  // Allocate a Particle structure
  *particles = (Particle*)malloc(sizeof(Particle));
  if (!*particles) {
    TERMINATE("Could not allocate particle array.\n");
  }

  // Allocate all of the Particle data arrays
  Particle* particle = *particles;
  size_t allocation = 0;
  allocation += allocate_data(&particle->x, nparticles * 1.5);
  allocation += allocate_data(&particle->y, nparticles * 1.5);
  allocation += allocate_data(&particle->omega_x, nparticles * 1.5);
  allocation += allocate_data(&particle->omega_y, nparticles * 1.5);
  allocation += allocate_data(&particle->energy, nparticles * 1.5);
  allocation += allocate_data(&particle->weight, nparticles * 1.5);
  allocation += allocate_data(&particle->dt_to_census, nparticles * 1.5);
  allocation += allocate_data(&particle->mfp_to_collision, nparticles * 1.5);
  allocation += allocate_int_data(&particle->cellx, nparticles * 1.5);
  allocation += allocate_int_data(&particle->celly, nparticles * 1.5);

  // Initialise all of the particle data
  const int nthreads = NTHREADS;
  const int nblocks = ceil(nparticles / (double)NTHREADS);
  inject_particles_kernel<<<nblocks, nthreads>>>(
      local_nx, local_ny, pad, x_off, y_off, local_particle_left_off,
      local_particle_bottom_off, local_particle_width, local_particle_height,
      nparticles, dt, initial_energy, edgex, edgey, (*particles)->x,
      (*particles)->y, (*particles)->cellx, (*particles)->celly,
      (*particles)->omega_x, (*particles)->omega_y, (*particles)->energy,
      (*particles)->weight, (*particles)->dt_to_census,
      (*particles)->mfp_to_collision);

  return allocation;
}

// Sends a particle to a neighbour and replaces in the particle list
void send_and_mark_particle(const int destination, Particle* particle) {}

// Validates the results of the simulation
void validate(const int nx, const int ny, const char* params_filename,
              const int rank, double* energy_deposition_tally) {

  double* h_energy_deposition_tally;
  allocate_host_data(&h_energy_deposition_tally, nx * ny);
  copy_buffer(nx * ny, &energy_deposition_tally, &h_energy_deposition_tally,
              RECV);

  // Reduce the energy deposition tally locally
  double local_energy_tally = 0.0;
  for (int ii = 0; ii < nx * ny; ++ii) {
    local_energy_tally += h_energy_deposition_tally[ii];
  }

  // Finalise the reduction globally
  double global_energy_tally = reduce_all_sum(local_energy_tally);

  if (rank != MASTER) {
    return;
  }

  printf("\nFinal global_energy_tally %.15e\n", global_energy_tally);

  int nresults = 0;
  char* keys = (char*)malloc(sizeof(char) * MAX_KEYS * (MAX_STR_LEN + 1));
  double* values = (double*)malloc(sizeof(double) * MAX_KEYS);
  if (!get_key_value_parameter(params_filename, NEUTRAL_TESTS, keys, values,
                               &nresults)) {
    printf("Warning. Test entry was not found, could NOT validate.\n");
    return;
  }

  // Check the value is within tolerance
  printf("Expected %.12e, result was %.12e.\n", values[0], global_energy_tally);
  if (within_tolerance(values[0], global_energy_tally, VALIDATE_TOLERANCE)) {
    printf("PASSED validation.\n");
  } else {
    printf("FAILED validation.\n");
  }

  free(keys);
  free(values);
}
